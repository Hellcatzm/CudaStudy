#include<hip/hip_runtime.h>
#include<stdio.h>

__device__ float devData;

__global__ void checkGlobalVariable(){
    printf("Device: the value of the global variable is %f\n", devData);
    devData += 2.0f;
}

int main(){
    float value = 3.14f;
    // cudaMemcpyToSymbol(devData, &value, sizeof(float));
    float *dptr = NULL;
    hipGetSymbolAddress((void**)&dptr, HIP_SYMBOL(devData));  // Symbol不是地址,本句获取其全局地址
    hipMemcpy(dptr, &value, sizeof(float), hipMemcpyHostToDevice);
    printf("Host:   copied %f to the global variable\n", value);

    checkGlobalVariable<<<1, 1>>>();

    hipMemcpyFromSymbol(&value, HIP_SYMBOL(devData), sizeof(float));
    printf("Host:   the value changed by the kernel to %f\n", value);

    hipDeviceReset();
    return EXIT_SUCCESS;
}