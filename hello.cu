
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void){
    printf("Hello World from GPU!\n");
}

int main(void){
    // hello from CPU
    printf("Hello World from CPU!\n");
    helloFromGPU<<<1, 10>>>();
    hipDeviceReset();
    return 0;
}